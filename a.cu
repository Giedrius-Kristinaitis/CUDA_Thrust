#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <iomanip>
#include "nlohmann/json.hpp"
#include ""

using namespace std;
using namespace nlohmann;

struct Product {
	char title[256];
	double price;
	int qty;
};

void readData(string file, vector<Product> *products);
void printResults(string fileName, Product* results, int resultCount);
__global__ void sum(Product* products, Product* results, int* rangeLength, int* productCount);
__device__ void concatString(char* destination, char* source);
__device__ void resetString(char* str);

// entry point of the program
int main() {
	vector<Product> products;
	
	readData("./IFF-7-2_Giedrius_Kristinaitis_L3_dat.json", &products);
	
	// host data
	Product* productPtr = &products[0];
	int productCount = products.size();
	int threadCount = 8;
	int rangeLength = products.size() / threadCount;
	Product* resultsPtr = new Product[threadCount];
	
	// device data
	Product* d_products;
	Product* d_results;
	int* d_rangeLength;
	int* d_productCount;
	
	// allocate device memory
	hipMalloc((void**) &d_products, productCount * sizeof(Product));
	hipMalloc((void**) &d_results, threadCount * sizeof(Product));
	hipMalloc((void**) &d_rangeLength, sizeof(int));
	hipMalloc((void**) &d_productCount, sizeof(int));
	
	// copy host data to device memory
	hipMemcpy(d_products, productPtr, productCount * sizeof(Product), hipMemcpyHostToDevice);
	hipMemcpy(d_rangeLength, &rangeLength, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_productCount, &productCount, sizeof(int), hipMemcpyHostToDevice);
	
	// perform calculations
	sum<<<1, threadCount>>>(d_products, d_results, d_rangeLength, d_productCount);

	// wait for other threads to finish
	hipDeviceSynchronize();
	
	// copy results to host
	hipMemcpy(resultsPtr, d_results, threadCount * sizeof(Product), hipMemcpyDeviceToHost);
	
	// print results
	printResults("./IFF-7-2_Giedrius_Kristinaitis_L3_Rez_A.txt", resultsPtr, threadCount);
	
	// free memory
	hipFree(d_products);
	hipFree(d_results);
	hipFree(d_rangeLength);
	hipFree(d_productCount);
	free(resultsPtr);
	
	return 0;
}

// performs calculations
__global__ void sum(Product* products, Product* results, int* rangeLength, int* productCount) {
	int startIndex = threadIdx.x * *rangeLength;
    int endIndex = min(startIndex + *rangeLength, *productCount);

	Product result;
	
	resetString(result.title);
	result.price = 0;
	result.qty = 0;
	
	for (int i = startIndex; i < endIndex; i++) {
		Product current = products[i];
		
		result.price += current.price;
		result.qty += current.qty;
		concatString(result.title, current.title);
	}
	
	results[threadIdx.x] = result;
}

// concats to strings
__device__ void concatString(char* destination, char* source) {
	for (int i = 0; i < 256; i++) {
		if (destination[i] == 0) {
			for (int j = 0; j < 256; j++) {
				if (source[j] == 0 || i + j > 255) {
					break;
				}
				
				destination[i + j] = source[j];
			}
			
			break;
		}
	}
}

// resets a string
__device__ void resetString(char* str) {
    for (int i = 0; i < 256; i++) {
        str[i] = 0;
    }
}

// prints results to a file
void printResults(string fileName, Product* results, int resultCount) {
	ofstream file;
	
    file.open(fileName, ios_base::out);
	
    file << setw(70) 
		 << "Results"
		 << endl
         << "--------------------------------------------------------------------------------------------------------------------------------------------"
         << endl
         << setw(100) 
		 << "Title |" 
		 << setw(10) 
		 << "Price |" 
		 << setw(10) 
		 << "Quantity" 
		 << endl
         << "--------------------------------------------------------------------------------------------------------------------------------------------"
         << endl;
		 
    for (int i = 0; i < resultCount; ++i) {
        string str(results[i].title);
		
        file << setw(100) 
			 << str
			 << " |"
			 << setw(10) 
			 << to_string(results[i].price)
			 << " |"
             << setw(10)
             << to_string(results[i].qty) 
			 << endl;
    }

    file << endl << endl << endl;
}

// reads product data from a file
void readData(string file, vector<Product> *products) {
	ifstream input;
    input.open(file, ifstream::in);

    json data = json::parse(input);

    for (auto& element: data["products"]) {
        Product product;
		
		strcpy(product.title, element.value("title", "").c_str());
		
		product.price = element.value("price", 0);
		product.qty = element.value("quantity", 0);

        products->push_back(product);
    }

    input.close();
}