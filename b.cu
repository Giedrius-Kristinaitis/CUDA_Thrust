#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <iomanip>
#include "nlohmann/json.hpp"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

using namespace std;
using namespace nlohmann;
using namespace thrust;

__device__ void concatString(char* destination, char* source);
void resetString(char* str);

struct Product {
	char title[256];
	double price;
	int qty;
	
	__device__ Product operator()(Product a, Product b) {
		a.price += b.price;
		a.qty += b.qty;
	
		concatString(a.title, b.title);
		
		return a;
	}
};

void readData(string file, host_vector<Product> *products);
void printResults(string fileName, Product* results);

int main() {
	// read data
	host_vector<Product> data(25);
	
	readData("./IFF-7-2_Giedrius_Kristinaitis_L3_dat.json", &data);
	
	// copy host data to device data
	device_vector<Product> d_data = data;

	// create initial data
	Product initial;
	
	resetString(initial.title);
	initial.price = 0;
	initial.qty = 0;
	
	// perform reduce
	Product result = reduce(d_data.begin(), d_data.end(), initial, Product());
	
	// print results
	printResults("./IFF-7-2_Giedrius_Kristinaitis_L3_Rez_B.txt", &result);
	
	return 0;
}

// concats to strings
__device__ void concatString(char* destination, char* source) {
	for (int i = 0; i < 256; i++) {
		if (destination[i] == 0) {
			for (int j = 0; j < 256; j++) {
				if (source[j] == 0 || i + j > 255) {
					break;
				}
				
				destination[i + j] = source[j];
			}
			
			break;
		}
	}
}

// resets a string
void resetString(char* str) {
    for (int i = 0; i < 256; i++) {
        str[i] = 0;
    }
}

// prints results to a file
void printResults(string fileName, Product* results) {
	ofstream file;
	
    file.open(fileName, ios_base::out);
	
    file << setw(70) 
		 << "Results"
		 << endl
         << "--------------------------------------------------------------------------------------------------------------------------------------------"
         << endl
         << setw(100) 
		 << "Title |" 
		 << setw(10) 
		 << "Price |" 
		 << setw(10) 
		 << "Quantity" 
		 << endl
         << "--------------------------------------------------------------------------------------------------------------------------------------------"
         << endl;
		 
    string str(results[0].title);
		
    file << setw(100) 
		 << str
		 << " |"
		 << setw(10) 
		 << to_string(results[0].price)
		 << " |"
         << setw(10)
         << to_string(results[0].qty) 
		 << endl;

    file << endl << endl << endl;
}

// reads product data from a file
void readData(string file, host_vector<Product> *products) {
	ifstream input;
    input.open(file, ifstream::in);

    json data = json::parse(input);

    for (auto& element: data["products"]) {
        Product product;
		
		strcpy(product.title, element.value("title", "").c_str());
		
		product.price = element.value("price", 0);
		product.qty = element.value("quantity", 0);

        products->push_back(product);
    }

    input.close();
}

